#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"


void gpu_histogram(int * hist_out, unsigned char * img_in, int img_size, int nbr_bin){
    int i;
    for ( i = 0; i < nbr_bin; i ++){
        hist_out[i] = 0;
    }

    for ( i = 0; i < img_size; i ++){
        hist_out[img_in[i]] ++;
    }
}

__global__ void gpu_histogram_equalization(unsigned char * img_out, unsigned char * img_in,
                            int * hist_in, int img_size, int nbr_bin, int numOfThreads, int * lut){


    int i = 0;
    int x = threadIdx.x + blockDim.x*blockIdx.x;

    int start;
    int end;
    //hist_in[x%256] = x;
    /* Get the result image */
    if(x >= img_size) {
       return;
    }
    start = ((img_size/numOfThreads) * x);
    if(numOfThreads == 1) {
       end = (img_size/numOfThreads);
    }
    else {
       end = ((img_size/numOfThreads) * (x+1));
    }
    for(i = start; i < end; i ++){
        if(lut[img_in[i]] > 255){
            img_out[i] = 255;
        }
        else{
            img_out[i] = (unsigned char)lut[img_in[i]];
        }
        
    }
}


